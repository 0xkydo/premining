#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <unistd.h>
#include <hip/hip_runtime.h>
#include <sys/time.h>
#include <pthread.h>
#include <locale.h>
#include "b2s.cuh"

/************************ MACROS *****************************/
// CUDA
#define THREADS 1024
#define BLOCKS 256
#define GPUS 3
// Mining
#define DIFFICULTY 4
#define RANDOM_LEN 64
// Files
#define MAX_FILE_LEN 1000

/************************ TYPES *****************************/

struct HandlerInput
{
    int device;
    unsigned long hashesProcessed;
    int prefix_len;
    int suffix_len;
};
typedef struct HandlerInput HandlerInput;

/************************ CONSTANTS *****************************/

__constant__ BYTE characterSet[17] = {"0123456789abcdef"};

/************************ DEVICE - HELPER *****************************/

__global__ void initSolutionMemory(char *blockContainsSolution)
{
    *blockContainsSolution = -1;
}

__device__ unsigned long deviceRandomGen(unsigned long x)
{
    x ^= (x << 21);
    x ^= (x >> 35);
    x ^= (x << 4);
    return x;
}

__device__ void digest_to_hex(char *hash, unsigned char digest[32])
{
    int hash_index = 0;

    for (int i = 0; i < 32; i++)
    {
        hash[hash_index++] = characterSet[(digest[i] & 0xF0) >> 4];
        hash[hash_index++] = characterSet[digest[i] & 0x0F];
    }

    hash[hash_index] = '\0';

    return;
}

/************************ DEVICE - MAIN *****************************/

__global__ void blake2s_cuda_mining(BYTE *prefix, int *prefix_len,
                                    BYTE *suffix, int *suffix_len,
                                    BYTE *nonce, char *hash,
                                    char *blockContainsSolution,
                                    unsigned long baseSeed)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    blake2s_state ctx;
    BYTE digest[32];
    BYTE random[RANDOM_LEN];
    unsigned long seed = baseSeed;
    seed += (unsigned long)i;
    for (int j = 0; j < 64; j++)
    {
        seed = deviceRandomGen(seed);
        int randomIdx = (int)(seed % 16);
        random[j] = characterSet[randomIdx];
    }

    // Initilize
    blake2s_init_device(&ctx);
    // Add in different parts of the input
    blake2s_update_device(&ctx, prefix, *prefix_len);
    blake2s_update_device(&ctx, random, RANDOM_LEN);
    blake2s_update_device(&ctx, suffix, *suffix_len);
    // Produce final hash
    blake2s_final_device(&ctx, digest);

    for (int j = 0; j < DIFFICULTY; j++)
        if (digest[j] > 0)
            return;
    if (digest[DIFFICULTY] > 0xAA)
    {
        return;
    }

    if (*blockContainsSolution == 1)
        return;
    *blockContainsSolution = 1;
    for (int j = 0; j < RANDOM_LEN; j++)
        nonce[j] = random[j];

    digest_to_hex(hash, digest);
}

/************************ HOST - HELPER *****************************/

void hostRandomGen(unsigned long *x)
{
    *x ^= (*x << 21);
    *x ^= (*x >> 35);
    *x ^= (*x << 4);
}

long long timems()
{
    struct timeval end;
    gettimeofday(&end, NULL);
    return end.tv_sec * 1000LL + end.tv_usec / 1000;
}

int read_file_into_array(const char *filename, unsigned char *array)
{
    FILE *fp;
    int length;

    fp = fopen(filename, "rb");
    if (fp == NULL)
    {
        printf("Error opening %s\n", filename);
        return -1;
    }

    length = fread(array, 1, MAX_FILE_LEN, fp);
    fclose(fp);

    return length;
}

/************************ HOST - VAR *****************************/

pthread_mutex_t solutionLock;
BYTE *nonce;
char *hash;

BYTE cpuPrefix[MAX_FILE_LEN];
BYTE cpuSuffix[MAX_FILE_LEN];

/************************ HOST - MAIN *****************************/

void *launchGPUHandlerThread(void *vargp)
{
    HandlerInput *hi = (HandlerInput *)vargp;
    hipSetDevice(hi->device);

    // HOST MEMORY ALLOCATION

    // Store nonce and solution;
    BYTE *nonceSolution = (BYTE *)malloc(sizeof(BYTE) * RANDOM_LEN);
    char *hashSolution = (char *)malloc(sizeof(char) * 65);
    char *blockContainsSolution = (char *)malloc(sizeof(char));
    unsigned long rngSeed = timems();

    // DEVICE MEMORY ALLOCATION

    // Store prefix
    BYTE *d_prefix;
    hipMalloc(&d_prefix, hi->prefix_len);
    hipMemcpy(d_prefix, cpuPrefix, hi->prefix_len, hipMemcpyHostToDevice);

    // Store suffix
    BYTE *d_suffix;
    hipMalloc(&d_suffix, hi->suffix_len);
    hipMemcpy(d_suffix, cpuSuffix, hi->suffix_len, hipMemcpyHostToDevice);

    // Store lengths
    int *d_prefix_len;
    hipMalloc(&d_prefix_len, sizeof(int));
    hipMemcpy(d_prefix_len, &(hi->prefix_len), sizeof(int), hipMemcpyHostToDevice);

    int *d_suffix_len;
    hipMalloc(&d_suffix_len, sizeof(int));
    hipMemcpy(d_suffix_len, &(hi->suffix_len), sizeof(int), hipMemcpyHostToDevice);

    BYTE *d_nonce;
    hipMalloc(&d_nonce, sizeof(BYTE) * RANDOM_LEN);

    char *d_hash;
    hipMalloc(&d_hash, sizeof(char) * 65);

    char *d_blockContainsSolution;
    hipMalloc(&d_blockContainsSolution, sizeof(char));

    initSolutionMemory<<<1, 1>>>(d_blockContainsSolution);

    while (1)
    {
        hostRandomGen(&rngSeed);

        hi->hashesProcessed += THREADS * BLOCKS;
        blake2s_cuda_mining<<<THREADS, BLOCKS>>>(d_prefix, d_prefix_len,
                                                 d_suffix, d_suffix_len,
                                                 d_nonce, d_hash,
                                                 d_blockContainsSolution,
                                                 rngSeed);
        hipDeviceSynchronize();

        hipMemcpy(blockContainsSolution, d_blockContainsSolution, sizeof(char), hipMemcpyDeviceToHost);
        if (*blockContainsSolution == 1)
        {
            // Copy Nonce
            hipMemcpy(nonceSolution, d_nonce, sizeof(BYTE) * RANDOM_LEN, hipMemcpyDeviceToHost);
            nonce = nonceSolution;
            // Copy Hash of the block
            hipMemcpy(hashSolution, d_hash, sizeof(char) * 65, hipMemcpyDeviceToHost);
            hash = hashSolution;

            pthread_mutex_unlock(&solutionLock);
            break;
        }

        if (nonce)
        {
            break;
        }
    }

    hipDeviceReset();
    return NULL;
}

int main()
{
    setlocale(LC_NUMERIC, "");
    pthread_mutex_init(&solutionLock, NULL);
    pthread_mutex_lock(&solutionLock);

    // Read in files.
    int prefix_len = read_file_into_array("prefix.txt", cpuPrefix);
    int suffix_len = read_file_into_array("suffix.txt", cpuSuffix);

    unsigned long **processedPtrs = (unsigned long **)malloc(sizeof(unsigned long *) * GPUS);
    pthread_t *tids = (pthread_t *)malloc(sizeof(pthread_t) * GPUS);
    long long start = timems();

    for (int i = 0; i < GPUS; i++)
    {
        HandlerInput *hi = (HandlerInput *)malloc(sizeof(HandlerInput));
        hi->device = i;
        hi->hashesProcessed = 0;
        hi->prefix_len = prefix_len;
        hi->suffix_len = suffix_len;
        processedPtrs[i] = &hi->hashesProcessed;
        pthread_create(tids + i, NULL, launchGPUHandlerThread, hi);
        usleep(10);
    }

    while (1)
    {
        // unsigned long totalProcessed = 0;
        // for (int i = 0; i < GPUS; i++)
        // {
        //     totalProcessed += *(processedPtrs[i]);
        // }
        // long long elapsed = timems() - start;

        // // printf("Hashes (%'lu) Seconds (%'f) Hashes/sec (%'lu)\r", totalProcessed, ((float)elapsed) / 1000.0, (unsigned long)((double)totalProcessed / (double)elapsed) * 1000);
        if (nonce)
        {
            break;
        }
    }
    //  printf("\n");

    pthread_mutex_lock(&solutionLock);
    //     long long end = timems();
    // long long elapsed = end - start;

    // Solution Found

    for (int i = 0; i < GPUS; i++)
    {
        pthread_join(tids[i], NULL);
    }

    // unsigned long totalProcessed = 0;
    // for (int i = 0; i < GPUS; i++)
    // {
    //     totalProcessed += *(processedPtrs[i]);
    // }

    // Change RANDOM_LEN here as well.
    printf("%.64s\n", nonce);
    printf("%.64s\n", hash);
    // printf("Difficulty: %i\n", DIFFICULTY);
    // printf("Hashes processed: %'lu\n", totalProcessed);
    // printf("Time: %llu ms\n", elapsed);
    // printf("Hashes/sec: %'lu\n", (unsigned long)((double)totalProcessed / (double)elapsed) * 1000);

    return 0;
}
